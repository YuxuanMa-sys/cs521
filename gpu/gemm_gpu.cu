#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < M; i++) { \
		for (int j = 0; j < N; j++) { \
			d_C_INI_ ## name[i * N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	// dim3 blockSize(1);
	// dim3 gridSize(1);
	// gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < M && j < N) {
		float sum = 0.0f;
		for (int k = 0; k < K; k++) {
			sum += A[i * K + k] * B[k * N + j];
		}
		C[i * N + j] = sum;
	}
}
void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{

	dim3 blockSize(16, 16);

	dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
				  (M + blockSize.y - 1) / blockSize.y);

	// Launch the kernel
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

#define TILE_SIZE 16
__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {
	__shared__ float tileA[TILE_SIZE][TILE_SIZE];
	__shared__ float tileB[TILE_SIZE][TILE_SIZE];

	int row = blockIdx.y * TILE_SIZE + threadIdx.y;
	int col = blockIdx.x * TILE_SIZE + threadIdx.x;

	float sum = 0.0f;

	for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; t++) {
		if (row < M && (t * TILE_SIZE + threadIdx.x) < K)
			tileA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
		else
			tileA[threadIdx.y][threadIdx.x] = 0.0f;

		if (col < N && (t * TILE_SIZE + threadIdx.y) < K)
			tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
		else
			tileB[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		for (int i = 0; i < TILE_SIZE; i++) {
			sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
		}

		__syncthreads();
	}

	if (row < M && col < N)
		C[row * N + col] = sum;
}
void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
	dim3 blockSize(TILE_SIZE, TILE_SIZE);
	dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE,
				  (M + TILE_SIZE - 1) / TILE_SIZE);
	gemm_gpu_o2_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

#define TILE_SIZE_OPT 16

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
	__shared__ float tileA[TILE_SIZE_OPT][TILE_SIZE_OPT];
	__shared__ float tileB[TILE_SIZE_OPT][TILE_SIZE_OPT];

	int row = blockIdx.y * TILE_SIZE_OPT + threadIdx.y;
	int col = blockIdx.x * TILE_SIZE_OPT + threadIdx.x;
	float sum = 0.0f;

	for (int t = 0; t < (K + TILE_SIZE_OPT - 1) / TILE_SIZE_OPT; t++) {
		if (row < M && t * TILE_SIZE_OPT + threadIdx.x < K)
			tileA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE_OPT + threadIdx.x];
		else
			tileA[threadIdx.y][threadIdx.x] = 0.0f;

		if (col < N && t * TILE_SIZE_OPT + threadIdx.y < K)
			tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE_OPT + threadIdx.y) * N + col];
		else
			tileB[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		#pragma unroll
		for (int i = 0; i < TILE_SIZE_OPT; i++) {
			sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
		}
		__syncthreads();
	}

	if (row < M && col < N)
		C[row * N + col] = sum;
}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(TILE_SIZE_OPT, TILE_SIZE_OPT);
	dim3 gridSize((N + TILE_SIZE_OPT - 1) / TILE_SIZE_OPT,
				  (M + TILE_SIZE_OPT - 1) / TILE_SIZE_OPT);
	gemm_gpu_o3_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}



int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}